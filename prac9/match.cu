//
// Pattern-matching program
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

////////////////////////////////////////////////////////////////////////
// include kernel function
////////////////////////////////////////////////////////////////////////

#include <match_kernel.cuh>

////////////////////////////////////////////////////////////////////////
// declare Gold routine
////////////////////////////////////////////////////////////////////////

void gold_match(unsigned int *, unsigned int *, int *, int, int);

////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////

int main(int argc, const char **argv){

  char *ctext, *cwords[] = {"cuti", "gold", "text", "word"};
  unsigned int  *text,  *words;

  int   length, len, nwords=4, matches[4]={0, 0, 0, 0};

  // read in text for processing

  FILE *fp;
  fp = fopen("match.cu","r");

  length = 0;
  while (getc(fp) != EOF) length++;

  ctext = (char *) malloc(length+4);

  rewind(fp);

  for (int l=0; l<length; l++) ctext[l] = getc(fp);
  for (int l=length; l<length+4; l++) ctext[l] = ' ';

  fclose(fp);

  // define number of words of text, and set pointers

  len  = length/4;
  text = (unsigned int *) ctext;

  // define words for matching

  words = (unsigned int *) malloc(nwords*sizeof(unsigned int));

  for (int w=0; w<nwords; w++) {
    words[w] = ((unsigned int) cwords[w][0])
             + ((unsigned int) cwords[w][1])*256
             + ((unsigned int) cwords[w][2])*256*256
             + ((unsigned int) cwords[w][3])*256*256*256;
  }

  ////////////////////////////////////////////////////////////////////////////////////////////////////
  // declare, allocate, and initialize device arrays
  unsigned int  *d_text, *d_words;
  int *d_matches;
  checkCudaErrors( hipMalloc((void**)&d_text, (length+4)*sizeof(char)) );
  checkCudaErrors( hipMalloc((void**)&d_words, nwords*sizeof(unsigned int)) );
  checkCudaErrors( hipMalloc((void**)&d_matches, nwords*sizeof(int)) );
  checkCudaErrors( hipMemcpy(d_text, text, (length+4)*sizeof(char), hipMemcpyHostToDevice));
  checkCudaErrors( hipMemcpy(d_words, words, nwords*sizeof(unsigned int), hipMemcpyHostToDevice));
  checkCudaErrors( hipMemset(d_matches, 0, nwords) );
  ////////////////////////////////////////////////////////////////////////////////////////////////////

  // CPU execution

  gold_match(text, words, matches, nwords, len);

  printf(" CPU matches = %d %d %d %d \n",
         matches[0],matches[1],matches[2],matches[3]);

  
  ////////////////////////////////////////////////////////////////////////////////////////////////////
  // GPU execution
  int bin_size = 16;
  dim3 blockDim(int((len-1) / bin_size) + 1);
  dim3 threadDim(bin_size, nwords);
  int shared_mem_size = bin_size * 4 + nwords; // bin_size * n_offsets * nwords
  match_kernel<<<blockDim, threadDim, shared_mem_size>>>(d_matches, d_text, d_words, len, nwords);
  checkCudaErrors( hipMemcpy(matches, d_matches, nwords*sizeof(int), hipMemcpyDeviceToHost) );
  printf(" GPU matches = %d %d %d %d \n",
         matches[0],matches[1],matches[2],matches[3]);
  ////////////////////////////////////////////////////////////////////////////////////////////////////


  // Release GPU and CPU memory


  free(ctext);
  free(words);
  checkCudaErrors( hipFree(d_text) );
  checkCudaErrors( hipFree(d_words) );
  checkCudaErrors( hipFree(d_matches) );
  // CUDA exit -- needed to flush printf write buffer
  hipDeviceReset();
}
